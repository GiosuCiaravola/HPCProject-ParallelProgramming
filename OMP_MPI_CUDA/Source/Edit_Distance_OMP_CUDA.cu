/*
 * Course: High Performance Computing 2023/2024
 *
 * Lecturer: Francesco Moscato	fmoscato@unisa.it
 *
 * Author:
 * Ciaravola Giosuè		0622702177		g.ciaravola3@studenti.unisa.it
 *
 * Copyright (C) 2024 - All Rights Reserved
 *
 * This file is part of ProjectHPC.
 *
 * This program is free software: you can redistribute it and/or modify it under the terms of
 * the GNU General Public License as published by the Free Software Foundation, either version
 * 3 of the License, or (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful, but WITHOUT ANY WARRANTY;
 * without even the implied warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.
 * See the GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License along with ContestOMP.
 * If not, see <https://www.gnu.org/licenses/gpl-3.0.html>.
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <string.h>
#include <omp.h>
#include <time.h>

#define CUDA_CHECK(X) {\
 hipError_t _m_cudaStat = X;\
 if(hipSuccess != _m_cudaStat) {\
    fprintf(stderr,"\nCUDA_ERROR: %s in file %s line %d\n",\
    hipGetErrorString(_m_cudaStat), __FILE__, __LINE__);\
    exit(1);\
 } }

/////////////////////////////////////////////////////////RANDOM STRING GENERATION/////////////////////////////////////////////////////////////////

/**
 * @brief Generates a random string of specified length using the given seed.
 *
 * @param n_characters The desired length of the generated string.
 * @param seed         The seed value for the random number generator.
 * 
 * @return             A dynamically allocated char array (string) containing
 *                     random characters. It is the responsibility of the caller
 *                     to free the allocated memory using free() when done using
 *                     the generated string.
 *                     Returns NULL in case of memory allocation failure.
 */
char* generateRandomString(int n_characters, int seed) {
    // Define the character alphabet for generating random strings
    static const char alphabet[] = "abcde fghijklmn opqrst uvwxyz";
    
    // Calculate the size of the alphabet (excluding the null terminator)
    int alphabetSize = sizeof(alphabet) - 1;

    // Allocate memory for the generated string, including space for the null terminator
    char* generatedString = (char*)malloc((n_characters + 1) * sizeof(char));

    // Check for memory allocation failure
    if (generatedString == NULL) {
        fprintf(stderr, "ERROR in memory allocation.\n");
        exit(EXIT_FAILURE);
    }

    // Seed the random number generator
    srand(seed);

    // Generate random characters based on the alphabet and fill the string
    for (int i = 0; i < n_characters; ++i)
        generatedString[i] = alphabet[rand() % alphabetSize];

    // Add the null terminator at the end of the generated string
    generatedString[n_characters] = '\0';

    // Return the generated random string
    return generatedString;
}

///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

/**
 * @brief CUDA function to compute the length of a null-terminated string.
 *
 * This function calculates the length of the input null-terminated string.
 *
 * @param str Pointer to the null-terminated input string.
 * @return    Length of the input string (excluding the null terminator).
 */
__host__ __device__ int cuda_strlen(const char* str) {
    int length = 0;
    while (str[length] != '\0') {
        length++;
    }
    return length;
}

/**
 * @brief CUDA function to copy a null-terminated string up to a specified size.
 *
 * This function copies characters from the source string to the destination
 * string up to the specified size. If the source string is shorter than the
 * specified size, the destination string is null-terminated.
 *
 * @param dest Pointer to the destination string.
 * @param src  Pointer to the source null-terminated string.
 * @param size Maximum number of characters to copy.
 */
__host__ __device__ void cuda_strcpy(char* dest, const char* src, int size) {
    for (int i = 0; i < size; ++i) {
        dest[i] = src[i];
        if (src[i] == '\0') {
            break;
        }
    }
}

/**
 * @brief CUDA function to find the minimum of two floating-point numbers.
 *
 * This function returns the minimum value between the two input floating-point numbers.
 *
 * @param a First floating-point number.
 * @param b Second floating-point number.
 * @return  The minimum of the two input floating-point numbers.
 */
__host__ __device__ float cuda_fmin(float a, float b) {
    return (a < b) ? a : b;
}

/**
 * @brief Calculates the partitioned substring based on the given parameters.
 *
 * This function takes the entire string, the index of the partition, and the total number
 * of partitions as input parameters. It calculates the appropriate substring for the specified
 * partition, attempting to distribute the elements as evenly as possible, and returns it as a
 * dynamically allocated char array.
 *
 * @param str   The entire string to be partitioned.
 * @param id    The index of the partition.
 * @param n_id  The total number of partitions.
 *
 * @return      A dynamically allocated char array containing the partitioned substring.
 *              It is the responsibility of the caller to free the allocated memory using free().
 */
__host__ __device__ char* partitioning(const char* str, int id, int n_id) {

    int len = cuda_strlen(str);

    // Calculate the lengths and starting points of local sections of work
    int local_len = len / n_id; 
    int remaining = len % n_id; 
    int local_start = id * local_len + cuda_fmin(id, remaining); 
    local_len += (id < remaining) ? 1 : 0; 

    // Allocate memory for the local substring
    char* local_str = (char*)malloc((local_len + 1) * sizeof(char));

    // Copy the partitioned substring into the allocated memory
    cuda_strcpy(local_str, str + local_start, local_len);
    local_str[local_len] = '\0';

    return local_str;
}

/**
 * @brief Calculates the Levenshtein distance between two strings.
 *
 * The Levenshtein distance is the minimum number of single-character edits 
 * (insertions, deletions, or substitutions) required to change one word into another.
 *
 * @param str1 The first input string.
 * @param str2 The second input string.
 * 
 * @return     The Levenshtein distance between the two input strings.
 *             Returns -1 if memory allocation fails during distance matrix creation.
 */
__host__ __device__ int levenshteinDistance(const char* str1, const char* str2) {
    // Lengths of input strings plus one for the null terminator
    int m = cuda_strlen(str1) + 1;
    int n = cuda_strlen(str2) + 1;

    // Allocate memory for the distance matrix
    int* distance = (int*)malloc(m * n * sizeof(int));

    // Populate the distance matrix using the Levenshtein distance algorithm
    for (int i = 0; i < m; ++i) {
        for (int j = 0; j < n; ++j) {
            if (i == 0) {
                distance[i * n + j] = j;
            } else if (j == 0) {
                distance[i * n + j] = i;
            } else if (str1[i - 1] == str2[j - 1]) {
                distance[i * n + j] = distance[(i - 1) * n + (j - 1)];
            } else {
                distance[i * n + j] = 1 + cuda_fmin(cuda_fmin(distance[(i - 1) * n + j], distance[i * n + (j - 1)]), distance[(i - 1) * n + (j - 1)]);
            }
        }
    }

    // Store the result and free the allocated memory
    int result = distance[m * n - 1];
    free(distance);

    return result;
}

/**
 * @brief CUDA kernel for computing the edit distance between two strings.
 *
 * This CUDA kernel divides the input strings among threads and computes the
 * Levenshtein distance locally for each thread. The results are stored in the
 * output array ED.
 *
 * @param str1 Pointer to the first input string.
 * @param str2 Pointer to the second input string.
 * @param ED   Pointer to the output array to store the computed edit distances.
 */
__global__ void EditDistanceKernel(char* str1, char* str2, int* ED) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int n_threads = blockDim.x*gridDim.x;

    // Partition the input strings for the current thread
    char* local_str1 = partitioning(str1, tid, n_threads);
    char* local_str2 = partitioning(str2, tid, n_threads);

    // Compute the Levenshtein distance locally for the current thread
    int local_ED = levenshteinDistance(local_str1, local_str2);

    // Store the local edit distance result in the output array
    ED[tid] = local_ED;
}

/**
 * @brief Perform Levenshtein distance calculation in a parallelized manner using OpenMP.
 *
 * This function partitions the input strings into multiple sections based on the number
 * of threads and calculates the Levenshtein distance for each section concurrently using
 * OpenMP parallelism. The results are combined to obtain the final distance.
 *
 * @param str1       The first input string.
 * @param str2       The second input string.
 * @param n_threads  Number of OpenMP threads.
 *
 * @return           The Levenshtein distance calculated in a parallelized manner.
 */
int threadPartitioning(const char* str1, const char* str2, int n_threads) {

    int* results = (int*)malloc(n_threads * sizeof(int));

    #pragma omp parallel
    {
        int tid = omp_get_thread_num();
        char* local_str1 = partitioning(str1, tid, n_threads);
        char* local_str2 = partitioning(str2, tid, n_threads);

        results[tid] = levenshteinDistance(local_str1, local_str2);

        free(local_str1);
        free(local_str2);
    }

    int local_result = 0;
    for(int i = 0; i < n_threads; i++)
        local_result += results[i];

    return local_result;
}

/** LEGGIMI LEGGIMI LEGGIMI LEGGIMI LEGGIMI LEGGIMI
 * @brief Appends timing information to a CSV file.
 *
 * This function takes the number of characters used in the program, the optimization used,
 * the timing for string generation, the timing for Kernel execution, 
 * the timing for overall program execution, the number of OpenMP threads used as input parameters.
 * (The number of CUDA threads is fixed)
 * It then appends the information to a CSV file in a formatted string.
 *
 * @param n_characters       Number of characters used in the program.
 * @param opt                Optimization level used in the program.
 * @param create_time        Time taken for string generation.
 * @param edit_distance_time Time taken for Edit Distance calculation.
 * @param execution_time     Total execution time.
 * @param omp_threads        Number of OpenMP threads used.
 * @param mpi_process        Number of MPI processes used.
 */
void printTimeToCSV(int n_characters, int opt, double create_time, double kernel_execution_time, double execution_time, int omp_threads) {
    // Define the file path
    char path[200];
    sprintf(path, "Informations/OMP_CUDA/opt%d/%d.csv", opt, n_characters); 
    char *filename = path;

    // Open the file for appending
    FILE *fp = fopen(filename, "a+");

    // Check if the file can be opened
    if (fp == NULL) {
        perror("ERROR during the file opening.");
        fprintf(stderr, "File %s can not be opened.\n", filename);
        return;
    }

    #ifdef L1_CACHE
        fprintf(fp, "OMP+CUDA_L1;%d;1024;%06f;%06f;%06f;\n", omp_threads, create_time, kernel_execution_time, execution_time);
    #else
        fprintf(fp, "OMP+CUDA;%d;1024;%06f;%06f;%06f;\n", omp_threads, create_time, kernel_execution_time, execution_time);
    #endif

    // Close the file
    fclose(fp);
}

/**
 * @brief Appends the Edit Distance OMP_CUDA result to a CSV file in a formatted string.
 *
 * This function takes the number of characters used in the program, the optimization used,
 * the result of the Edit Distance, the number of OpenMP threads used as input parameters.
 * (The number of CUDA threads is fixed) 
 * It then appends the information to a CSV file in a formatted string.
 *
 * @param n_characters The number of characters used in the program.
 * @param opt          The type of optimization used.
 * @param distance     The result of the Edit Distance calculation.
 * @param omp_threads  The number of OpenMP threads used.
 */
void printResultToCSV(int n_characters, int opt, int distance, int omp_threads) {
    // Define the file path
    char path[200];
    sprintf(path, "EditDistanceReport/OMP_CUDA/opt%d/%d.csv", opt, n_characters); 
    char *filename = path;

    // Open the file for appending
    FILE *fp = fopen(filename, "a+");
    
    // Check if the file can be opened
    if (fp == NULL) {
        perror("ERROR during the file opening.");
        fprintf(stderr, "File %s cannot be opened.\n", filename);
        return;
    }

    // Change the print if is an execution with the L1 cache.
    #ifdef L1_CACHE
        fprintf(fp, "OMP+CUDA_L1;%d;1024;%d\n", omp_threads, distance);
    #else
        fprintf(fp, "OMP+CUDA;%d;1024;%d\n", omp_threads, distance);
    #endif

    // Close the file
    fclose(fp);
}

/**
 * @brief Perform Edit Distance computation using both OpenMP and CUDA.
 *
 * This function divides the workload between OpenMP and CUDA, computes the Edit Distance
 * between two input strings, and returns the elapsed time for the Kernel computation.
 *
 * @param str1       Pointer to the first input string.
 * @param str2       Pointer to the second input string.
 * @param n_threads  Number of OpenMP threads to use.
 * @param opt        Optimization level used in the program.
 * @return           Elapsed time for the Kernel computation.
 */
float editDistanceOnDevice(char* str1, char* str2, int n_threads, int opt) {
    // Divide the input strings for both GPU and host processing
    char* temp_gpu_str1 = partitioning(str1, 0, 2);
    int size_gpu_str1 = strlen(temp_gpu_str1);
    char* host_str1 = partitioning(str1, 1, 2);

    char* temp_gpu_str2 = partitioning(str2, 0, 2);
    int size_gpu_str2 = strlen(temp_gpu_str2);
    char* host_str2 = partitioning(str2, 1, 2);

    // Allocate and copy GPU memory
    char* gpu_str1;
    char* gpu_str2;
    CUDA_CHECK(hipMalloc((void**)&gpu_str1, size_gpu_str1*sizeof(char)));
    CUDA_CHECK(hipMalloc((void**)&gpu_str2, size_gpu_str2*sizeof(char)));
    CUDA_CHECK(hipMemcpy(gpu_str1, temp_gpu_str1, size_gpu_str1*sizeof(char),hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(gpu_str2, temp_gpu_str2, size_gpu_str2*sizeof(char),hipMemcpyHostToDevice));

   // Calculate the number of GPU threads blocks
    int n_grid = size_gpu_str1/20; //Just to make sure that each thread takes a few extra characters
    dim3 blockSize = 1024;
    dim3 gridSize = ((n_grid)/blockSize.x + 1);
    int cuda_threads = gridSize.x * blockSize.x;

    // Allocate GPU memory for Edit Distance results
    int* gpu_ED;
    CUDA_CHECK(hipMalloc((void**)&gpu_ED, cuda_threads * sizeof(int)));

    // Create CUDA events for timing
    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));

    // Set cache configuration if optimization is enabled
    #ifdef L1_CACHE
        cudaFuncSetCacheConfig(EditDistanceKernel,cudaFuncCachePreferL1); //Function to prioritize the L1 cache to 48KB
    #endif

    // Record the start time and execute the CUDA kernel
    CUDA_CHECK(hipEventRecord(start,0));
    EditDistanceKernel<<<gridSize, blockSize>>>(gpu_str1, gpu_str2, gpu_ED);
    CUDA_CHECK(hipEventRecord(stop,0));

    // Perform thread partitioning on the host
    int final_ED = threadPartitioning(host_str1, host_str2, n_threads);

    // Synchronize GPU
    CUDA_CHECK(hipDeviceSynchronize());

    // Copy results back to host, and calculate final Edit Distance
    int* result_gpu_ED = (int*)malloc(cuda_threads * sizeof(int));
    CUDA_CHECK(hipMemcpy(result_gpu_ED,gpu_ED,cuda_threads * sizeof(int),hipMemcpyDeviceToHost));

    for(int i = 0; i < cuda_threads; i++) {
        final_ED += result_gpu_ED[i];
    }

    // Calculate elapsed time
    float elapsed;
    CUDA_CHECK(hipEventElapsedTime(&elapsed,start,stop));
    elapsed = elapsed/1000.f;

    // Cleanup GPU resources
    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));
    CUDA_CHECK(hipFree(gpu_str1));
    CUDA_CHECK(hipFree(gpu_str2));
    CUDA_CHECK(hipFree(gpu_ED));

    // Print result and free memory
    int n_characters = strlen(str1);
    printResultToCSV(n_characters, opt, final_ED, n_threads);

    free(temp_gpu_str1);
    free(temp_gpu_str2);
    free(host_str1);
    free(host_str2);
    free(result_gpu_ED);

    return elapsed;
}

/**
 * @brief Main function for launching the Edit Distance computation program.
 *
 * This function generates random strings, performs Edit Distance computation
 * using both OpenMP and CUDA, and prints the execution time to a CSV file.
 *
 * @param argc Number of command line arguments.
 * @param argv Array of command line arguments.
 *             Usage: <program_name> <n_characters> <seed1> <seed2> <opt> <n_threads>
 * @return     0 if the program executes successfully, 1 otherwise.
 */
int main(int argc, char* argv[]) {

    // Check if the correct number of command line arguments is provided
    if (argc != 6) {
        fprintf(stderr, "Usage: %s <n_characters> <seed1> <seed2> <opt> <n_threads>\n", argv[0]);
        return 1;
    }

    // Parse command line arguments
    int n_characters = atoi(argv[1]);
    int seed1 = atoi(argv[2]);
    int seed2 = atoi(argv[3]);
    int opt = atoi(argv[4]);
    int n_threads = atoi(argv[5]);
    
    // Set the number of OpenMP threads
    omp_set_num_threads(n_threads);

    // Variables for tracking execution time
    double program_execution, create_time;
    float kernel_execution_time;
    
    // Record the start time for the entire program
    clock_t start_execution = clock();
    // Record the start time for string creation
    clock_t start_creation = clock();

    // Generate random strings
    char* str1 = generateRandomString(n_characters, seed1);
    char* str2 = generateRandomString(n_characters, seed2);

    // Record the end time for string creation
    clock_t end_creation = clock();

    // Calculate and store the time taken for string creation
    create_time = (double)(end_creation - start_creation) / CLOCKS_PER_SEC;

    // Perform Edit Distance computation and record the kernel execution time
    kernel_execution_time = editDistanceOnDevice(str1, str2, n_threads, opt);

    // Free memory allocated for strings
    free(str1);
    free(str2);

    // Record the end time for the entire program
    clock_t end_execution = clock();
    // Calculate and store the total program execution time
    program_execution = (double)(end_execution - start_execution) / CLOCKS_PER_SEC;

    // Print the execution times to a CSV file
    printTimeToCSV(n_characters, opt, create_time, kernel_execution_time, program_execution, n_threads);

    return 0;
}